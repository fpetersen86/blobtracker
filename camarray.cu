#include "hip/hip_runtime.h"
#include "camarray.h"
#include <stdio.h>
#include "webcamtest.h"
#include "global.h"

#ifdef CUDA
#include "utility_environment.h"
#include <hip/hip_runtime.h>


//GPU Kernel
__global__ void lensCorrection(char *image, char *output, int width, int height, int width2, int height2, float strength, float zoom)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;         // coordinates within 2d array follow from block index and thread index within block
	int y = blockIdx.y * blockDim.y + threadIdx.y;
    int elemID = y*width2 + x;                              // index within linear array
    
    x += (width-width2)/2;
	y += (height-height2)/2;
	
	int halfWidth = width / 2;
	int halfHeight = height / 2;
	float correctionRadius = sqrtf(width * width + height * height) / strength;
	int newX = x - halfWidth;
	int newY = y - halfHeight;

	float distance = sqrtf(newX * newX + newY * newY);
	float r = distance / correctionRadius;
	
	float theta;
	if(r != 0)
	{
		theta = atanf(r)/r;
	} else {
		theta = 1;
	}
	
	int sourceX = halfWidth + theta * newX * zoom;
	int sourceY = halfHeight + theta * newY * zoom;
	
	output[elemID] = image[sourceY*width + sourceX];
}

__global__ void lensCorrection2(char *image, char *output, int width, int height, int width2, int height2, float strength, float zoom)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;         // coordinates within 2d array follow from block index and thread index within block
	int y = blockIdx.y * blockDim.y + threadIdx.y;
    int elemID = y*width + x;                              // index within linear array
    
	output[elemID] = image[elemID];
}

#endif


CamArray::CamArray(webcamtest* p) : QThread(p)
{
	w = p; // All hail the mighty alphabet ;)

}

#ifdef CUDA
void CamArray::initPlatform() {
	//host buffers
	hipHostMalloc(&h_a, bufferSize);
	hipHostMalloc(&h_b, bufferSize2);
	
	//device buffers
	hipMalloc((void**) &d_a, bufferSize);
	hipMalloc((void**) &d_b, bufferSize2);
}
#else //NOCUDA
void CamArray::initPlatform() {
	h_a = reinterpret_cast<char*>(malloc(bufferSize));
	h_b = reinterpret_cast<char*>(malloc(bufferSize2));
}
#endif


void CamArray::run()
{
	stopped = false;
	
	//initialise cams
	QDir d("/dev/");
	d.setFilter(QDir::System);
	d.setNameFilters(QStringList("*video*"));
	QStringList camList = d.entryList();
	numCams = camList.size();
	QString c;
	QSemaphore *sem = new QSemaphore(numCams);
	sem->acquire(numCams);
	
	bufferSize = xSize * ySize * numCams * sizeof(char);
	bufferSize2 = xSize2 * ySize2 * numCams * sizeof(char);
	
	initPlatform();
	
	for(int i = 0; i < numCams; i++)
	{
		c = camList.at(i);
		cams[i] = new Camera(d.absoluteFilePath(c).toStdString().c_str(), i, sem, h_a);
	}
	
	//start capturing
	for(int i = 0; i < numCams; i++)
	{
		cams[i]->start();
	}
	
	mainloop();
}

#ifdef CUDA
void CamArray::mainloopCUDA()
{
	dim3 cudaBlockSize(16,16);  // image is subdivided into rectangular tiles for parallelism - this variable controls tile size
	dim3 cudaGridSize(xSize2/cudaBlockSize.x, ySize2/cudaBlockSize.y);
	
	while(!stopped)
	{
		sem->acquire(numCams);
		hipMemcpy( d_a, h_a, bufferSize, hipMemcpyHostToDevice );
		handleCUDAerror(__LINE__);
		
		lensCorrection<<<cudaGridSize, cudaBlockSize>>>(d_a, d_b, xSize, ySize, xSize2, ySize2, lcStrength, lcZoom);
		handleCUDAerror(__LINE__);
		
		hipMemcpy( h_b, d_b, bufferSize2, hipMemcpyDeviceToHost );
		handleCUDAerror(__LINE__);
		
		output();
	}
}

#endif

#ifdef NOCUDA
void CamArray::mainloopCPU()
{
	while(!stopped)
	{
		sem->acquire(numCams);
		
		int width = xSize;
		int height = ySize;
		int width2 = xSize2;
		int height2 = ySize2;
		float strength = lcStrength;
		float zoom = lcZoom;
		
		for (int y = 0; y < ySize2; y++)
		{
			for (int x = 0; x < xSize2; x++)
			{
				int myX = x;
				int myY = y;
				int elemID = myY*width2 + myX;                              // index within linear array

				myX += (width-width2)/2;
				myY += (height-height2)/2;
				
				int halfWidth = width / 2;
				int halfHeight = height / 2;
				float correctionRadius = sqrt(width * width + height * height) / strength;
				int newX = myX - halfWidth;
				int newY = myY - halfHeight;

				float distance = sqrt(newX * newX + newY * newY);
				float r = distance / correctionRadius;
				
				float theta;
				if(r != 0)
				{
					theta = atan(r)/r;
				} else {
					theta = 1;
				}
				
				int sourceX = halfWidth + theta * newX * zoom;
				int sourceY = halfHeight + theta * newY * zoom;
				
				h_b[elemID] = h_a[sourceY*width + sourceX];
				//qDebug("elemID: %d   X: %d myX: %d sourceX: %d   Y: %d myY: %d sourceY: %d", elemID, x, myX, sourceX, y, myY, sourceY);
			}
		}
		output();
	}
}
#endif

void CamArray::output()
{
	for (int y = 0; y < ySize; y++)
	{
		for (int x = 0; x < xSize; x++)
		{
			int val = h_a[y*xSize+x];
			w->i.setPixel(x,y, qRgb(val, val, val));
		}
	}
	
	for (int y = 0; y < ySize2; y++)
	{
		for (int x = 0; x < xSize2; x++)
		{
			int val = h_b[y*xSize2+x];
			w->i.setPixel(x,y+ySize, qRgb(val, val, val));
		}
	}
	w->update();
	//qDebug("available: %d", sem->available());
}

void CamArray::stop()
{
	stopped = true;
}


void CamArray::loadFile(QString filenName)
{
	QImage fileImage(filenName);

	for (int y = 0; y < fileImage.height(); y++)
	{
		for (int x = 0; x < fileImage.width(); x++)
		{
			h_a[y*fileImage.width()+x] = qGray(fileImage.pixel(x,y));
		}
	}
}

CamArray::~CamArray()
{
	for (int i = 0; i < numCams; i++)
	{
		cams[i]->stop();
	}
	for (int i = 0; i < numCams; i++)
	{
		cams[i]->wait();
		delete cams[i];
	}
	qDebug() << "CamArray stopped";
	
	// free memory buffers
#ifndef NOCUDA
	hipFree(d_a);
	handleCUDAerror(__LINE__);
	hipFree(d_b);
	handleCUDAerror(__LINE__);
	hipHostFree(h_a);
	handleCUDAerror(__LINE__);
	hipHostFree(h_b);
	handleCUDAerror(__LINE__);
#else
	free(h_a);
	free(h_b);
#endif
	qDebug("Memory deallocated successfully\n");
}

