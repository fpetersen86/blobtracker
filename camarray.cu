#include "hip/hip_runtime.h"
#include "camarray.h"
#include <stdio.h>
#include "webcamtest.h"
#include "global.h"

#ifdef CUDA
#include "utility_environment.h"
#include <hip/hip_runtime.h>


//GPU Kernel
__global__ void lensCorrection(char *image, char *output, int width, int height, int width2, int height2, float strength, float zoom)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;         // coordinates within 2d array follow from block index and thread index within block
	int y = blockIdx.y * blockDim.y + threadIdx.y;
    int elemID = y*width2 + x + blockIdx.z*blockDim.x*blockDim.y;                             // index within linear array
    
    x += (width-width2)/2;
	y += (height-height2)/2;
	
	int halfWidth = width / 2;
	int halfHeight = height / 2;
	float correctionRadius = sqrtf(width * width + height * height) / strength;
	int newX = x - halfWidth;
	int newY = y - halfHeight;

	float distance = sqrtf(newX * newX + newY * newY);
	float r = distance / correctionRadius;
	
	float theta;
	if(r != 0)
	{
		theta = atanf(r)/r;
	} else {
		theta = 1;
	}
	
	int sourceX = halfWidth + theta * newX * zoom;
	int sourceY = halfHeight + theta * newY * zoom;
	
	output[elemID] = image[sourceY*width + sourceX];
}

__global__ void lensCorrection2(char *image, char *output, int width, int height, int width2, int height2, float strength, float zoom)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;         // coordinates within 2d array follow from block index and thread index within block
	int y = blockIdx.y * blockDim.y + threadIdx.y;
    int elemID = y*width + x;                              // index within linear array
    
	output[elemID] = image[elemID];
}

#endif


CamArray::CamArray(webcamtest* p, int testimages) : QThread(p)
{
	w = p; // All hail the mighty alphabet ;)
	//initialise cams
	QStringList camList;
	QDir d("/dev/");
	if (!testimages)
	{
		imgTest = false;
		d.setFilter(QDir::System);
		d.setNameFilters(QStringList("*video*"));
		camList = d.entryList();
		numCams = camList.size();
	}
	else
	{
		imgTest = true;
		numCams = testimages;
	}
	p->resizeImage(numCams);
	QString c;
	sem = new QSemaphore(numCams);
	sem->acquire(numCams);
	
	bufferSize = xSize * ySize * numCams * sizeof(char);
	bufferSize2 = xSize2 * ySize2 * numCams * sizeof(char);
	threshold = 20;
	
	initBuffers();
	
	if (!imgTest)
		for(int i = 0; i < numCams; i++)
		{
			c = camList.at(i);
			cams[i] = new Camera(d.absoluteFilePath(c).toStdString().c_str(), i, sem, h_a);
		}
}

#ifdef CUDA
void CamArray::initBuffers() {
	//host buffers
	hipHostMalloc(&h_a, bufferSize);
	hipHostMalloc(&h_b, bufferSize2);
	hipHostMalloc(&h_c, bufferSize2);
	
	//device buffers
	hipMalloc((void**) &d_a, bufferSize);
	hipMalloc((void**) &d_b, bufferSize2);
	hipMalloc((void**) &d_c, bufferSize2);
}
#else //NOCUDA
void CamArray::initBuffers() {
	h_a = reinterpret_cast<char*>(malloc(bufferSize));
	h_b = reinterpret_cast<char*>(malloc(bufferSize2));
	h_c = reinterpret_cast<char*>(malloc(bufferSize2));
}
#endif


void CamArray::run()
{
	stopped = false;
	
	//start capturing
	if (!imgTest)
		for(int i = 0; i < numCams; i++)
		{
			cams[i]->start();
		}
	
	mainloop();
}

#ifdef CUDA
void CamArray::mainloop()
{
	dim3 cudaBlockSize(16,16);  // image is subdivided into rectangular tiles for parallelism - this variable controls tile size
	dim3 cudaGridSize(xSize2/cudaBlockSize.x, ySize2/cudaBlockSize.y, numCams);
	
	while(!stopped)
	{
		if (!imgTest)
			sem->acquire(numCams);
		else 
			msleep(8);
		hipMemcpy( d_a, h_a, bufferSize, hipMemcpyHostToDevice );
		handleCUDAerror(__LINE__);
		
		lensCorrection<<<cudaGridSize, cudaBlockSize>>>(d_a, d_b, xSize, ySize, xSize2, ySize2, lcStrength, lcZoom);
		handleCUDAerror(__LINE__);
		
		hipMemcpy( h_b, d_b, bufferSize2, hipMemcpyDeviceToHost );
		handleCUDAerror(__LINE__);
		
		output();
	}
}

#endif

#ifdef NOCUDA
void CamArray::mainloop()
{
	while(!stopped)
	{
		if (!imgTest)
			sem->acquire(numCams);
		else 
			msleep(8);
		
		int width = xSize;
		int height = ySize;
		int width2 = xSize2;
		int height2 = ySize2;
		float strength = lcStrength;
		float zoom = lcZoom;
		int myX, myY;
		
		int offset = 0;
		// lensCorrection
		for( int n = 0; n < numCams; n++)
		{
			for (int y = 0; y < ySize2; y++)
			{
				for (int x = 0; x < xSize2; x++)
				{
					myX = x;
					myY = y;
					int elemID = myY*width2 + myX;                              // index within linear array

					myX += (width-width2)/2;
					myY += (height-height2)/2;
					
					int halfWidth = width / 2;
					int halfHeight = height / 2;
					float correctionRadius = sqrt(width * width + height * height) / strength;
					int newX = myX - halfWidth;
					int newY = myY - halfHeight;

					float distance = sqrt(newX * newX + newY * newY);
					float r = distance / correctionRadius;
					
					float theta;
					if(r != 0)
					{
						theta = atan(r)/r;
					} else {
						theta = 1;
					}
					
					int sourceX = halfWidth + theta * newX * zoom;
					int sourceY = halfHeight + theta * newY * zoom;
					
					h_b[offset + elemID] = h_a[offset + sourceY*width + sourceX];
					//qDebug("elemID: %d   X: %d myX: %d sourceX: %d   Y: %d myY: %d sourceY: %d", elemID, x, myX, sourceX, y, myY, sourceY);
				}
			}
			offset   += xSize*ySize;
		}
		
		// rotate
		float sin_, cos_;
		offset = 0;
		int ydiff, xdiff;
		int xCenter = width/2, yCenter = height/2;
		
		for( int n = 0; n < numCams; n++)
		{
			sin_ = sin(cams[n]->angle);
			cos_ = cos(cams[n]->angle);
			for (int y = 0; y < ySize2; y++)
			{
				ydiff = yCenter - y;
				for (int x = 0; x < xSize2; x++)
				{
					xdiff = xCenter - x;
					myX = xCenter + (-xdiff * cos_ - ydiff * sin_);
					if (myX < 0 || myX >= width)
						continue;
					myY = yCenter + (-ydiff * cos_ + xdiff * sin_);
					if (myY < 0 || myY >= height)
						continue;
					h_c[offset + y*width + x] = h_b[offset + myY*width + myX];
				}
			}
			offset += ySize*xSize;
		}
		
		
		output();
	}
}
#endif


void CamArray::output()
{
	int offset = 0, xOffset = 0, xOffset2 = 0;
	for( int n = 0; n < numCams; n++)
	{
		for (int y = 0; y < ySize; y++)
		{
			for (int x = 0; x < xSize; x++)
			{
				int val = h_a[offset+y*xSize+x];
				w->i.setPixel(xOffset+x,y, qRgb(val, val, val));
			}
		}

		for (int y = 0; y < ySize2; y++)
		{
			for (int x = 0; x < xSize2; x++)
			{
				int val = h_b[offset+y*xSize2+x];
				w->i.setPixel(xOffset2+x,y+ySize, qRgb(val, val, val));
			}
		}
		for (int y = 0; y < ySize2; y++)
		{
			for (int x = 0; x < xSize2; x++)
			{
				int val = h_c[offset+y*xSize2+x];
				w->i.setPixel(xOffset2+x,y+ySize*2, qRgb(val, val, val));
			}
		}
		for (int y = 0; y < ySize2; y++)
		{
			for (int x = 0; x < xSize2; x++)
			{
				int val = h_c[offset+y*xSize2+x] <= threshold ? 255 : 0;
				w->i.setPixel(xOffset2+x,y+ySize*3, qRgb(val, val, val));
			}
		}
		xOffset  += xSize;
		xOffset2 += xSize2;
		offset   += xSize*ySize;
	}
	w->update();
	//qDebug("available: %d", sem->available());
}

void CamArray::stop()
{
	stopped = true;
}


void CamArray::loadFiles()
{
	QImage fileImage;
	int offset = 0;
	for (int j = 1; j < QApplication::arguments().size(); j++)
	{
		fileImage = QImage(QApplication::arguments().at(j)).scaled(xSize, ySize,
												Qt::IgnoreAspectRatio, Qt::SmoothTransformation);
		qDebug() << QApplication::arguments().at(j) << " xsize " << fileImage.width() ;
		for (int y = 0; y < ySize; y++)
		{
			for (int x = 0; x < xSize; x++)
			{
				h_a[offset+y*xSize+x] = qGray(fileImage.pixel(x,y));
			}
		}
		offset += (ySize * xSize);
	}
	w->update();
}

CamArray::~CamArray()
{
	for (int i = 0; i < numCams; i++)
	{
		cams[i]->stop();
	}
	for (int i = 0; i < numCams; i++)
	{
		cams[i]->wait();
		delete cams[i];
	}
	qDebug() << "CamArray stopped";
	
	// free memory buffers
#ifndef NOCUDA
	hipFree(d_a);
	handleCUDAerror(__LINE__);
	hipFree(d_b);
	handleCUDAerror(__LINE__);
	hipHostFree(h_a);
	handleCUDAerror(__LINE__);
	hipHostFree(h_b);
	handleCUDAerror(__LINE__);
#else
	free(h_a);
	free(h_b);
#endif
	qDebug("Memory deallocated successfully\n");
}

